
// splits with tree

#include <stdlib.h>
#include <stdio.h>
#include <png.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include<assert.h>
using namespace std;

__global__ void merge(int *mat,struct tree* t1,struct tree* c1,struct tree* c2,struct tree* c3,struct tree* c4,unsigned int *, unsigned int * );

int width, height;
png_byte color_type;
png_byte bit_depth;
png_bytep *row_pointers;

struct tree
{
	int start1,end1,start2,end2,data,label;
	int fg1,fg2,fg3,fg4; // to find the adjacencies
	struct tree *c1,*c2,*c3,*c4;
}*root;

struct region
{
	int x1,y1,x2,y2,x3,y3,x4,y4,mean;
};
	int w=0,h=0;
	int **mat;
	static int count;
	vector<region> childs;
	int read_png_file(char *);
	void write_png_file(char *);
	void process_png_file(unsigned int);
	bool pred(int , int ,int ,int ,int *mat[]);
	int mean(int,int,int ,int ,int *mat[]);
	region split(region,int *mat[],unsigned int, struct tree*);
	//void merge(int *mat[],struct tree* t1,int );
	__host__ __device__ bool mergeregion(struct tree* t1, struct tree* t2);
	void labelling(int *mat[],struct tree* t1,struct tree* t2);
	void mergeglobe(int *mat[], struct tree* t1,struct tree* t2,struct tree* t3, struct tree* t4);
	void print(struct tree*);
	int get_height(struct tree*);
	void printlevelorder(struct tree*, unsigned int);
	void printgivenlevel(struct tree*,int, unsigned int);


int read_png_file(char *filename) {
  FILE *fp = fopen(filename, "rb");

  png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  if(!png) abort();

  png_infop info = png_create_info_struct(png);
  if(!info) abort();

  if(setjmp(png_jmpbuf(png))) abort();

  png_init_io(png, fp);

  png_read_info(png, info);

  width      = png_get_image_width(png, info);
  height     = png_get_image_height(png, info);
  color_type = png_get_color_type(png, info);
  bit_depth  = png_get_bit_depth(png, info);

  if(bit_depth == 16)
    png_set_strip_16(png);

  if(color_type == PNG_COLOR_TYPE_PALETTE)
    png_set_palette_to_rgb(png);

  if(color_type == PNG_COLOR_TYPE_GRAY && bit_depth < 8)
    png_set_expand_gray_1_2_4_to_8(png);

  if(png_get_valid(png, info, PNG_INFO_tRNS))
    png_set_tRNS_to_alpha(png);

  if(color_type == PNG_COLOR_TYPE_RGB ||
     color_type == PNG_COLOR_TYPE_GRAY ||
     color_type == PNG_COLOR_TYPE_PALETTE)
    png_set_filler(png, 0xFF, PNG_FILLER_AFTER);

  if(color_type == PNG_COLOR_TYPE_GRAY ||
     color_type == PNG_COLOR_TYPE_GRAY_ALPHA)
    png_set_gray_to_rgb(png);

  png_read_update_info(png, info);

  row_pointers = (png_bytep*)malloc(sizeof(png_bytep) * height);
  for(int y = 0; y < height; y++) {
    row_pointers[y] = (png_byte*)malloc(png_get_rowbytes(png,info));
  }

  png_read_image(png, row_pointers);
  cout << "Height" << height << "\t Width" << width ;
  fclose(fp);
  int max;
  if(height > width)
  {
	max=height;
  }
  else
  {
	max=width;
  }
  int next = pow(2,ceil(log(max)/log(2)));
  cout << "\nNext \t" << next;
  return next;
  
}


void write_png_file(char *filename) {
  int y;

  FILE *fp = fopen(filename, "wb");
  if(!fp) abort();

  png_structp png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  if (!png) abort();

  png_infop info = png_create_info_struct(png);
  if (!info) abort();

  if (setjmp(png_jmpbuf(png))) abort();

  png_init_io(png, fp);


  png_set_IHDR(
    png,
    info,
    width, height,
    8,
    PNG_COLOR_TYPE_RGBA,
    PNG_INTERLACE_NONE,
    PNG_COMPRESSION_TYPE_DEFAULT,
    PNG_FILTER_TYPE_DEFAULT
  );
  png_write_info(png, info);

  png_write_image(png, row_pointers);
  png_write_end(png, NULL);

  for(int y = 0; y < height; y++) {
    free(row_pointers[y]);
  }
  free(row_pointers);

  fclose(fp);
}
//to find height of tree
int get_height(struct tree *t)
{
	int m,k;
	if(t==NULL)
		return 0;
	else
		{
			int c1h = get_height(t->c1);
			int c2h = get_height(t->c2);
			int c3h = get_height(t->c3);
			int c4h = get_height(t->c4);
			if(c1h>c2h)
			{
				m=c1h;
			}
			else
			{
				m=c2h;
			}
			if(c3h>c4h)
			{
				k=c3h;
			}
			else
			{
				k=c4h;
			}
			if(k>m)
			{
				return (k+1);
			}
			else
			{
				return (m+1);
			}

		}
	             

}
void printlevelorder(struct tree *root,unsigned int m1)
{
	int h = get_height(root);
	
	int i;
	for(i=h;i>=1;i--)
	{
		cout << "\ni: " << i;
		printgivenlevel(root,i,m1);
		
	}
}
unsigned int label1 = 5;

void printgivenlevel(struct tree *root,int level,unsigned int m1)
{
	//cout << "\nLevel : " <<  level ;
int *p = new int[m1*m1];
int *mat2 = new int[m1*m1];
unsigned int *da;
unsigned int *lab;
	if(root==NULL)
		return;
	if(level==1)
	{
		cout << "\nNode: \t(" << root->start1 << "," << root->end1 << ") (" << root->start2 << "," << root->end2 << ")" << "\tData: " << root->data << "\tFG: " << root->fg1 << root->fg2 << root->fg3 << root->fg4;
		//cout << "\nLevel : " <<  level ;
	}
	else if(level>1)
		{
			printgivenlevel(root->c1,level-1,m1);
			printgivenlevel(root->c2,level-1,m1);
			printgivenlevel(root->c3,level-1,m1);
			printgivenlevel(root->c4,level-1,m1);
			cout << "\n Merge ";
			//cout << "\nT1: " << root->fg2;
			
			if(root->c1!=NULL && root->c2!=NULL && root->c3!=NULL && root->c4!=NULL)
			{
			
			//cout << "\nD ";
			for(int h =0; h < m1; h++)
			{
				for(int w =0; w < m1; w++)
				{
					p[m1*h + w] = mat[h][w];
					//cout << "\t " << p[m1*h + m1];
				}
				//cout << "\n";
			}
			/*cout << "\nPREE:\n";
			for(int h =0; h < m1; h++)
			{	
				for(int w =0; w < m1; w++)
				{
					cout<<"\t"<<p[m1*h + w];
				}
				cout<<"\n";
			}  */

			struct tree * tree_d,*tree_c1,*tree_c2,*tree_c3,*tree_c4;	
			tree_d = new tree();
			tree_c1 = new tree();
			tree_c2 = new tree();
			tree_c3 = new tree();
			tree_c4 = new tree();
					
			hipMalloc((void **)&tree_d,  5* sizeof(struct  node*));
			hipMalloc((void **)&tree_c1, 5*sizeof(struct node*));
			hipMalloc((void **)&tree_c2, 5*sizeof(struct node*));
			hipMalloc((void **)&tree_c3, 5*sizeof(struct node*));
			hipMalloc((void **)&tree_c4, 5*sizeof(struct node*));			

			hipMalloc((void **)&da,sizeof(unsigned int));
			hipMalloc((void **)&mat2,sizeof(int)*m1*m1);
			hipMalloc((void **)&lab,sizeof(unsigned int));
			hipMemcpy(mat2,p,sizeof(int)*m1*m1,hipMemcpyHostToDevice);
			hipMemcpy(da,&m1,sizeof(unsigned int ),hipMemcpyHostToDevice);
			hipMemcpy(lab,&label1,sizeof(unsigned int ),hipMemcpyHostToDevice);	

			hipMemcpy(&(tree_d->start1),&( root->start1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_d->end1),&( root->end1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_d->start2),&( root->start2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_d->end2),&( root->end2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_d->data),&( root->data),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_d->label),&( root->label),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_d->fg1),&( root->fg1),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_d->fg2),&( root->fg2),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_d->fg3),&( root->fg3),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_d->fg4),&( root->fg4),sizeof(int), hipMemcpyHostToDevice);	

			hipMemcpy(&(tree_c1->start1),&( root->c1->start1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c1->end1),&( root->c1->end1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c1->start2),&( root->c1->start2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c1->end2),&( root->c1->end2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c1->data),&( root->c1->data),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c1->label),&( root->c1->label),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c1->fg1),&( root->c1->fg1),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c1->fg2),&( root->c1->fg2),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c1->fg3),&( root->c1->fg3),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c1->fg4),&( root->c1->fg4),sizeof(int), hipMemcpyHostToDevice);	

			hipMemcpy(&(tree_c2->start1),&( root->c2->start1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c2->end1),&( root->c2->end1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c2->start2),&( root->c2->start2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c2->end2),&( root->c2->end2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c2->data),&( root->c2->data),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c2->label),&( root->c2->label),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c2->fg1),&( root->c2->fg1),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c2->fg2),&( root->c2->fg2),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c2->fg3),&( root->c2->fg3),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c2->fg4),&( root->c2->fg4),sizeof(int), hipMemcpyHostToDevice);	

			hipMemcpy(&(tree_c3->start1),&( root->c3->start1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c3->end1),&( root->c3->end1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c3->start2),&( root->c3->start2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c3->end2),&( root->c3->end2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c3->data),&( root->c3->data),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c3->label),&( root->c3->label),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c3->fg1),&( root->c3->fg1),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c3->fg2),&( root->c3->fg2),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c3->fg3),&( root->c3->fg3),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c3->fg4),&( root->c3->fg4),sizeof(int), hipMemcpyHostToDevice);	

			hipMemcpy(&(tree_c4->start1),&( root->c4->start1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c4->end1),&( root->c4->end1),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c4->start2),&( root->c4->start2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c4->end2),&( root->c4->end2),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c4->data),&( root->c4->data),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c4->label),&( root->c4->label),sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(&(tree_c4->fg1),&( root->c4->fg1),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c4->fg2),&( root->c4->fg2),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c4->fg3),&( root->c4->fg3),sizeof(int), hipMemcpyHostToDevice);	
			hipMemcpy(&(tree_c4->fg4),&( root->c4->fg4),sizeof(int), hipMemcpyHostToDevice);	

			merge<<<1,4>>>(mat2,tree_d,tree_c1,tree_c2,tree_c3,tree_c4,lab,da);
			hipDeviceSynchronize();
			printf("\nSAM:");
			hipMemcpy(p,mat2,sizeof(int)*m1*m1,hipMemcpyDeviceToHost);
			/*
			for(int i = 0; i < m1; i++)
			{
				for(int j=0; j<m1; j++)
				{
					//mat[i][j] = mat1[m1*i + j];
					printf("%d\t",p[m1*i + j]);
				}
				printf("\n");
			}	*/
			for(int i = 0; i < m1; i++)
			{
				for(int j=0; j<m1; j++)
				{
					//mat[i][j] = mat1[m1*i + j];
					mat[i][j]=p[m1*i + j];
				}
				//printf("\n");
			}	
			//mergeglobe(mat,root->c1,root->c2,root->c3,root->c4);
			label1 = label1+4;
			hipFree(tree_d);
			hipFree(tree_c1);
			hipFree(tree_c2);
			hipFree(tree_c3);
			hipFree(tree_c4);			
			hipFree(da);
			hipFree(mat2);
			hipFree(lab);
		}
	
	}

}

//merge: 

__global__ void merge(int *mat1,struct tree* t1,struct tree* c1,struct tree* c2,struct tree* c3,struct tree* c4,unsigned int *label2,unsigned int *m2)
{
	//printf("\nkernel");
	t1->c1=c1;
	t1->c2=c2;
	t1->c3=c3;
	t1->c4=c4;	
	unsigned int m1 = *m2;
	unsigned int label1 = *label2;
	//printf("\nM1: %d",m1);
	int **mat = new int*[m1*m1];
	for( int i=0;i<m1;i++)
	{
		mat[i]=new int[m1];
	}
	for(int i = 0; i < m1; i++)
	{
		for(int j=0; j<m1; j++)
		{
			mat[i][j] = mat1[m1*i + j];
		}
	}
	/*printf("\nPRE\n");	
	for(int i = 0; i < m1; i++) 
	{
		for(int j=0; j<m1; j++)
		{
			printf("%d \t",mat[i][j]);
		}
		printf("\n");
	} */	
	bool row1=false,row2=false,col1=false,col2=false;
	if(t1->c1==NULL && t1->c2==NULL && t1->c3==NULL && t1->c4==NULL)
		return;

		row1 = mergeregion(t1->c1, t1->c2);
		row2 = mergeregion(t1->c3,t1->c4);
		col1 = mergeregion(t1->c1, t1->c3);	
		col2 = mergeregion(t1->c2, t1->c4);
	
	if( row1 == true )
	{
		for(int i=t1->c1->start1; i < t1->c1->start2; i++)
		{
			for(int j=t1->c1->end1; j < t1->c1->end2; j++)
			{
				if( mat[i][j] != 0)
				{
					mat[i][j] = label1;
				}
			}
		}
		//print
		/*printf("\nLocal merge ");
		for(int i=t1->c1->start1; i < t1->c1->start2; i++)
		{
			for(int j=t1->c1->end1; j < t1->c1->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/
		
		if( label1 > 0 )
			t1->c1->label = label1;
		t1->data = t1->c1->data;
		//printf("T1: %d" ,t1->data);
		for(int i=t1->c2->start1; i < t1->c2->start2; i++)
		{
			for(int j=t1->c2->end1; j < t1->c2->end2; j++)
			{
				if(mat[i][j] != 0)
				{
					mat[i][j] = label1;
				}
			}
		}
		//print
		/*printf("\nLocal merge ");
		for(int i=t1->c2->start1; i < t1->c2->start2; i++)
		{
			for(int j=t1->c2->end1; j < t1->c2->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/
		
		if( label1 > 0 )	
			t1->c2->label = label1;
		// take the data
		t1->data = t1->c2->data;
		//printf("T1: %d",t1->data);
	}
	
	if( row2 == true )
	{
		for(int i=t1->c3->start1; i < t1->c3->start2; i++)
		{
			for(int j=t1->c3->end1; j < t1->c3->end2; j++)
			{
				if(mat[i][j] != 0)
				{
					mat[i][j] = label1+1;
				}
			}
		}
		
		//print
		//cout << "\nLocal merge ";
		/*for(int i=t1->c3->start1; i < t1->c3->start2; i++)
		{
			for(int j=t1->c3->end1; j < t1->c3->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/

		//label1 = label1+1;
		if( label1 > 0 )	
			t1->c3->label = label1;
		t1->data = t1->c3->data;
		//printf("T1: %d",t1->data);
		for(int i=t1->c4->start1; i < t1->c4->start2; i++)
		{
			for(int j=t1->c4->end1; j < t1->c4->end2; j++)
			{
				if(mat[i][j] != 0)
				{
					mat[i][j] = label1+1;
				}
			}
		}
		

		//print
		//cout << "\nLocal merge ";
		/*for(int i=t1->c4->start1; i < t1->c4->start2; i++)
		{
			for(int j=t1->c4->end1; j < t1->c4->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/

		if( label1 > 0 )	
			t1->c4->label = label1;
		t1->data = t1->c4->data;
		//printf("T1: %d",t1->data);

	}
	//cout << "\tRow1 " << row1 << "\tRow2 " <<  row2 ;

	if( col1 == true )
	{
		if( row1 == true )
		{
			if( t1->c1->label > 0 )	
			{
				t1->c3->label = t1->c1->label;
				for(int i=t1->c3->start1; i < t1->c3->start2; i++)
				{	
					for(int j=t1->c3->end1; j < t1->c3->end2; j++)
					{
						if(mat[i][j] != 0)
						{
							mat[i][j] = t1->c1->label;
						}
					}
				}
			}	


			//print

			//cout << "\nLocal merge ";
			/*for(int i=t1->c3->start1; i < t1->c3->start2; i++)
			{
				for(int j=t1->c3->end1; j < t1->c3->end2; j++)
				{
					printf("%d\t",mat[i][j]);
				}
				printf("\n");
			}	*/
		
		t1->data = t1->c1->data;
		//printf("T1: %d",t1->data);
		}
		
		else
		{
			for(int i=t1->c1->start1; i < t1->c1->start2; i++)
			{
				for(int j=t1->c1->end1; j < t1->c1->end2; j++)
				{
					if(mat[i][j] != 0)
					{
						mat[i][j] = label1+1;
					}
				}
			}

			//print
			//cout << "\nLocal merge ";
		/*for(int i=t1->c1->start1; i < t1->c1->start2; i++)
		{
			for(int j=t1->c1->end1; j < t1->c1->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/			

			//label1 = label1+1;
			if( label1 > 0 )
				t1->c1->label = label1;
			t1->data = t1->c1->data;
			//cout << "T1: " << t1->data;
			for(int i=t1->c3->start1; i < t1->c3->start2; i++)
			{
				for(int j=t1->c3->end1; j < t1->c3->end2; j++)
				{
					if(mat[i][j] != 0)
					{
						mat[i][j] = label1+1;
					}
				}
			}
		

			//print
			//cout << "\nLocal merge ";
		/*for(int i=t1->c3->start1; i < t1->c3->start2; i++)
		{
			for(int j=t1->c3->end1; j < t1->c3->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/
		
			if( label1 > 0 )	
				t1->c3->label = label1;	
			t1->data = t1->c3->data;
			//printf("T1: %d" ,t1->data);
		}
	
	}
	
	if( col2 == true )
	{
		if( row2 == true )
		{
		if( t1->c2->label > 0 )	
		{
			t1->c4->label = t1->c2->label;
			for(int i=t1->c4->start1; i < t1->c4->start2; i++)
			{
				for(int j=t1->c4->end1; j < t1->c4->end2; j++)
				{
					if(mat[i][j] != 0)
					{
						mat[i][j] = t1->c2->label;
					}
				}
			}
		}
		//print
		//cout << "\nLocal merge ";
		/*for(int i=t1->c4->start1; i < t1->c4->start2; i++)
		{
			for(int j=t1->c4->end1; j < t1->c4->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/
		t1->data = t1->c2->data;
		//printf("T1: %d" ,t1->data);
		}
		
		else
		{
			//print
			//cout << "\nLocal merge ";
			/*for(int i=t1->c2->start1; i < t1->c2->start2; i++)
			{
				for(int j=t1->c2->end1; j < t1->c2->end2; j++)
				{
					printf("%d\t",mat[i][j]);
				}
			printf("\n");
			}	*/
	
				//label1 = label1+1;
				if( label1 > 0 )
					t1->c2->label = label1;
				t1->data = t1->c2->data;
				//printf("T1: %d",t1->data);
				for(int i=t1->c4->start1; i < t1->c4->start2; i++)
				{
					for(int j=t1->c4->end1; j < t1->c4->end2; j++)
					{
						if(mat[i][j] != 0)
						{
							mat[i][j] = label1;
						}
					}
				}

			//print
			//cout << "\nLocal merge ";
		/*for(int i=t1->c4->start1; i < t1->c4->start2; i++)
		{
			for(int j=t1->c4->end1; j < t1->c4->end2; j++)
			{
				printf("%d\t",mat[i][j]);
			}
			printf("\n");
		}	*/
		
			if( label1 > 0 )
				t1->c4->label = label1;
			t1->data = t1->c4->data;
			//printf("T1: %d",t1->data);
		}
	}
	/*printf("\nPOST\n");	
	for(int i = 0; i < m1; i++)
	{
		for(int j=0; j<m1; j++)
		{
			//mat[i][j] = mat1[m1*i + j];
			printf("%d\t",mat[i][j]);
		}
		printf("\n");
	}		*/
	for(int h =0; h < m1; h++)
			{
				for(int w =0; w < m1; w++)
				{
					mat1[m1*h + w] = mat[h][w];
					//cout << "\t " << p[m1*h + m1];
				}
				//cout << "\n";
			}
}

__host__ __device__ bool mergeregion(struct tree* t1, struct tree* t2)
{
	if(t1->data!=1 && t2->data!=1 && t1->data == t2->data)
	{ 
		cout << "\n\nMerging: T1 -> (" << t1->start1 << "\t" << t1->end1 << "),(" << t1->start2 << "\t" << t1->end2 << ")\tData " << t1->data <<"\t T2 -> (" << t2->start1 << "\t" << t2->end1 << "),(" << t2->start2 << "\t" << t2->end2 << ")\tData " << t2->data;
		return true;
	}			
	else
		return false;
}

void labelling(int *mat[],struct tree* t1,struct tree* t2)
{
	cout << "\n\nLblng: T1 -> (" << t1->start1 << "\t" << t1->end1 << "),(" << t1->start2 << "\t" << t1->end2 << ")\tData " << t1->data <<"\t T2 -> (" << t2->start1 << "\t" << t2->end1 << "),(" << t2->start2 << "\t" << t2->end2 << ")\tData " << t2->data << "  " << t2->label;
	

	for(int i=t1->start1; i < t1->start2; i++)
	{
		for(int j=t1->end1; j < t1->end2; j++)
		{
			if(mat[i][j] != 0 && t2->label > 0)
			{
				mat[i][j] = t2->label;
			}				
		}
		cout << "\n";
	}
	for(int i=t1->start1; i < t1->start2; i++)
	{
		for(int j=t1->end1; j < t1->end2; j++)
		{
			cout << "\t" << mat[i][j] ;			
		}
		cout << "\n";
	}		
	t1->label = t2->label;	
				
}
//global merge
void mergeglobe(int *mat[], struct tree* t1,struct tree* t2,struct tree* t3, struct tree* t4)
{
 // 1 -2 & 3 - 4
	if(t1!=NULL && t2!=NULL && t3!=NULL && t4!=NULL)
	{
		if(t1->fg2 == 1 && t2->fg1 == 1)
		{
			//if( t1->c2->data == t2->c1->data )
			//{
				if( t1->c2->label > t2->c1->label )
				{
					labelling( mat, t1->c2, t2->c1 );
				}
				else
				{
					labelling( mat, t2->c1, t1->c2 );	
				}
			//}		
		} 
		
		if(t1->fg4 == 1 && t2->fg3 == 1)
		{
			//if( t1->c4->data == t2->c3->data )
			//{
				if( t1->c4->label > t2->c3->label )
				{
					labelling( mat, t1->c4, t2->c3 );
				}
				else
				{
					labelling( mat, t2->c3, t1->c4 );
				}	
			//}		
		} 
		if(t3->fg2 == 1 && t4->fg1 == 1)
		{
			//if( t3->c2->data == t4->c1->data )
			//{
				if( t3->c2->label > t4->c1->label )
				{
					labelling( mat, t3->c2, t4->c1 );
				}
				else
				{
					labelling( mat, t4->c1, t3->c2 );	
				}
			//}		
		}
		if(t3->fg4 == 1 && t4->fg3 == 1)
		{
			//if( t3->c4->data == t4->c3->data )
			//{
				if( t3->c4->label > t4->c3->label )
				{
					labelling( mat, t3->c4, t4->c3 );
				}
				else
				{
					labelling( mat, t4->c3, t3->c4 );
				}	
			//}		
		}
// 3-1 & 4-2
		if(t1->fg3 == 1 && t3->fg1 == 1)
		{
			//if( t1->c3->data == t3->c1->data )
			//{
				if( t1->c3->label > t3->c1->label )
				{
					labelling( mat, t1->c3, t3->c1 );
				}
				else
				{
					labelling( mat, t3->c1, t1->c3 );	
				}
			//}		
		}
		if(t1->fg4 == 1 && t3->fg2 == 1)
		{
			//if( t1->c4->data == t3->c2->data )
			//{
				if( t1->c4->label > t3->c2->label )
				{
					labelling( mat, t1->c4, t3->c2 );
				}
				else
				{
					labelling( mat, t3->c2, t1->c4 );
				}	
			//}		
		}	
		if(t2->fg3 == 1 && t4->fg1 == 1)
		{	
			//if( t2->c3->data == t4->c1->data )
			//{
				if( t2->c3->label > t4->c1->label )
				{
					labelling( mat, t2->c3, t4->c1 );
				}
				else
				{
					labelling( mat, t4->c1, t2->c3 );	
				}
			//}		
		}
		if(t2->fg4 == 1 && t4->fg2 == 1)
		{
			//if( t2->c4->data == t4->c2->data )
			//{
				if( t2->c4->label > t4->c2->label )
				{
					labelling( mat, t2->c4, t4->c2 );
				}
				else
				{
					labelling( mat, t4->c2, t2->c4 );
				}	
			//}		
		}
		
	}						 			
}	
	
bool pred(int h1, int w1, int h,int w,int *mat[])
{
	int mean1 = mean(h1,w1,h,w,mat);
    	double var = 0; 
	int std_dev;
    	for (int a = h1; a < h; a++)
    	{
    		for (int b = w1; b < w; b++)
    		{
    			var += ((mat[a][b] - mean1) * (mat[a][b] - mean1));
    			
    		}
    	}
	int dx = h-h1;
	int dy = w-w1;
    	var /= (dx*dy);
	cout << "\nVar: " << var << "\t";
    	std_dev = sqrt(var);
	cout << "\nStddev: " << std_dev << "\t";
	return (std_dev <= 5.8) || ((dx*dy) <= 1) ;	
}

void print(struct tree* root1)
{
	 
	if(root1!=NULL)
	{
	//cout << "In: \n";
		if(root1->data!=-100 && root1->data!=1)
		{
		cout << "\nNode: (" << root1->start1 << "," << root1->end1 << ") (" << root1->start2 << "," << root1->end2 << ")" << "\tData: " << root1->data << "\tfg : " << root1->fg1 << root1->fg2 << root1->fg3 << root1->fg4;
		}
		else
		{
		
		}
		print(root1->c1);
		print(root1->c2);
		print(root1->c3);
		print(root1->c4);
	}
}


int  mean(int h1, int w1,int h,int w,int *mat[])
{
	double total = 0; int mean;
    	for (int i = h1; i < h; i++)
    	{
    		for (int j = w1; j < w; j++)
    		{
 
    			total += mat[i][j];
    		}
    	}
	int dx = h-h1;
	int dy = w-w1;
	cout << "\nTotal\t" <<total;
    	mean = (total/ (dx*dy));
	cout << "\nMean\t" << mean;
	return mean;	
}

void  process_png_file(unsigned int m1) {
 
mat=new int*[m1];
	for( int i=0;i<m1;i++)
	{
		mat[i]=new int[m1];
	}
 for(int y = 0; y < m1; y++) 
{
	for(int x = 0; x < m1; x++) 
	{
		mat[y][x]=0;
	} 
		printf("\n");
}

for(int y = 0; y < height; y++) 
{
	printf("\n");
	png_bytep row = row_pointers[y];
    	for(int x = 0; x < width; x++) 
	{
		png_bytep px = &(row[x * 4]);
      		//printf("RGB(%3d, %3d, %3d)\n",px[0], px[1], px[2]);  
      		int a = 0.72*px[0] + 0.72*px[1] + 0.72*px[2];
     		if( a > 128 )
      		{
      			mat[y][x]=a;
     		}
      		else
		{
			mat[y][x]=0;
		}
   	 }
}
printf("\nMatrix after thresholding\n");
for(int y = 0; y < m1; y++) 
{
	for(int x = 0; x < m1; x++) 
	{
		printf("%d\t",mat[y][x]);
	} 
	printf("\n");
} 

region r;
r.x1 = 0;
r.y1 = 0;
r.x4 = m1;
r.y4 = m1;

root = new tree();
struct tree *temp = new tree();
temp->start1 = r.x1;
temp->end1 = r.y1;
temp->start2 = r.x4;
temp->end2 = r.y4;
root=temp;

//Splitting :
split(r,mat,m1,temp);

//printing trees
//cout << "\nTRee before\n";
//print(root);
printf("\nMatrix after splitting \n");
for(int y = 0; y < m1; y++) 
{
	for(int x = 0; x < m1; x++) 
	{
		printf("%d\t",mat[y][x]);
	}
	printf("\n");
} 
cout << "\nLevel Order Traversal of Tree: \n" ;
printlevelorder(root,m1);

//cout << "\nTree after\n";
//print(root);
// Colour
int col = 5;
for(int y = 0; y < height; y++) 
{
    	png_bytep row = row_pointers[y];
    		for(int x = 0; x < width; x++) 
		{
			if( mat[y][x] != 0 )
			{
				png_bytep px = &(row[x * 4]);
				int mod = (mat[y][x]%5);
				if(mod == 0)
				{
					px[0]=50;
					px[1]=100;
					px[2]=150;
				}
				if(mod == 1)
				{
					px[0]=100;
					px[1]=200;
					px[2]=300;	
				}
				if(mod == 2) 
				{
					px[0]=200;
					px[1]=400;
					px[2]=600;	
				}
				if(mod == 3)
				{
					px[0]=400;
					px[1]=800;
					px[2]=1200;	
				}
				if(mod == 4)
				{
					px[0]=800;
					px[1]=1600;
					px[2]=2400;	
				}
    			}
		}
		
}
}
region split( region r ,int *mat[], unsigned int m1, struct tree *temp1)
{
//count++;

bool mean1=pred(r.x1,r.y1,r.x4,r.y4,mat);
int mean2 = mean(r.x1,r.y1,r.x4,r.y4,mat);
temp1->data = mean2;
if(mean1)
{
	
	cout << "\nLabelling (" << r.x1 << "\t" << r.y1 << ")\t(" << r.x4 << "\t" << r.y4 << ")\n" ;
	int mean2 = mean(r.x1,r.y1,r.x4,r.y4,mat);
	for( int i=r.x1; i < r.x4; i++)
	{
		for( int j=r.y1; j < r.y4; j++)
		{
			mat[i][j]=mean2;
		}
	}
	
	temp1->data = mean2;
	int p;
	for( p=0 ; p < childs.size() ; p++)
	{
		if(childs[p].x1 == r.x1 && childs[p].y1 == r.y1 && childs[p].x4 == r.x4 && childs[p].y4 == r.y4)
		{
			//cout << "\np: " << p;
			childs.erase(childs.begin() + p); 
			break;
		}
	}
}	
else
{
	count++;
	cout << "\nSplitting ("<< r.x1 << "\t" << r.y1 << ")\t(" << r.x4 << "\t" << r.y4 << ")\n" ;
	int w = ceil(m1/2);
	int h = ceil(m1/2);
	//r.size1=r.size1/2;
	region r1,r2,r3,r4;
	temp1->c1 = new tree();
	temp1->c2 = new tree();
	temp1->c3 = new tree();
	temp1->c4 = new tree();

	r1.x1 = r.x1,r1.y1 = r.y1,r1.x4 = r.x1+h,r1.y4 = r.y1+w;
	r2.x1 = r.x1,r2.y1 = r.y1+w,r2.x4 = r.x1+h,r2.y4 = r.y1+m1;
	r3.x1 = r.x1+h,r3.y1 = r.y1,r3.x4 = r.x1+m1,r3.y4 = r.y1+h;
	r4.x1 = r.x1+h,r4.y1 = r.y1+w,r4.x4 = r.x4,r4.y4 = r.y4;

	temp1->c1->start1 = r1.x1, temp1->c1->end1 = r1.y1, temp1->c1->start2 = r1.x4, temp1->c1->end2 = r1.y4;
	temp1->c2->start1 = r2.x1, temp1->c2->end1 = r2.y1, temp1->c2->start2 = r2.x4, temp1->c2->end2 = r2.y4;
	temp1->c3->start1 = r3.x1, temp1->c3->end1 = r3.y1, temp1->c3->start2 = r3.x4, temp1->c3->end2 = r3.y4;
	temp1->c4->start1 = r4.x1, temp1->c4->end1 = r4.y1, temp1->c4->start2 = r4.x4, temp1->c4->end2 = r4.y4;

	//find the means to set fg
	int m1 = mean(r1.x1,r1.y1,r1.x4,r1.y4,mat);
	int m2 = mean(r2.x1,r2.y1,r2.x4,r2.y4,mat);
	int m3 = mean(r3.x1,r3.y1,r3.x4,r3.y4,mat);
	int m4 = mean(r4.x1,r4.y1,r4.x4,r4.y4,mat);
	cout << "\nMeans : " << m1 << " " << m2 << " " << m3 << " " << m4 ;	
	if(m1 > 0)
	{
		temp1->fg1 = 1;	
	}
	if(m2 > 0)
	{
		temp1->fg2 = 1;	
	}
	if(m3 > 0)
	{
		temp1->fg3 = 1;	
	}
	if(m4 > 0)
	{
		temp1->fg4 = 1;	
	}

	childs.push_back(r1);
	childs.push_back(r2);
	childs.push_back(r3);
	childs.push_back(r4);
	/*cout << "\nVector after push : \n" ;
	cout << "\nVector size : " << childs.size() << "\n" ;
	for( int i=0 ;i < childs.size(); i++)
	{
		cout << "\t (" << childs[i].x1 << "," << childs[i].y1 << "),";
		cout << "(" << childs[i].x4 << "," << childs[i].y4 << ")";		
	}	*/
	//childs.erase(childs.begin());
	int p;
	for( p=0 ; p < childs.size() ; p++)
	{
		if(childs[p].x1 == r.x1 && childs[p].y1 == r.y1 && childs[p].x4 == r.x4 && childs[p].y4 == r.y4)
		{
			//cout << "\np: " << p;
			childs.erase(childs.begin() + p); 
			break;
		}
	}
	/*cout << "\nVector after erase : \n" ;
	for( int i=0 ;i < childs.size(); i++)
	{
		cout << "\t (" << childs[i].x1 << "," << childs[i].y1 << ")";
		cout << "(" << childs[i].x4 << "," << childs[i].y4 << ")";		
	} */
	r1=split(r1,mat,w,temp1->c1);
	r2=split(r2,mat,w,temp1->c2);
	r3=split(r3,mat,w,temp1->c3);
	r4=split(r4,mat,w,temp1->c4);
	
} 

/*cout << "\nVector size : " << childs.size() << "\n" ;
cout << "FG : " << temp1->fg1 << " " << temp1->fg2 << " " << temp1->fg3 << " " << temp1->fg4; */
}



int main(int argc, char *argv[]) {
  if(argc != 3) abort();
  		clock_t begin,end;
		double time_spent;
		begin=clock();
unsigned int m = read_png_file(argv[1]);
  cout << "\nM: " << m;
  process_png_file(m);
  write_png_file(argv[2]);
  cout<<"\n\nNo. of splits:\t"<<count;
  		end=clock();
		time_spent=(double)(end-begin)/CLOCKS_PER_SEC;
		printf("\nTIME : %lf",time_spent);
  cout<<"\nVector size:\n"<<childs.size();
  cout << "\nVector final: \n" ;
	for( int i=0 ;i < childs.size(); i++)
	{
		cout << "\t (" << childs[i].x1 << "," << childs[i].y1 << ")";
		cout << "(" << childs[i].x4 << "," << childs[i].y4 << ")";		
	}
	cout << "\nFINALE: ";
			for(int i = 0; i < m; i++)
			{
				for(int j=0; j<m; j++)
				{
					//mat[i][j] = mat1[m1*i + j];
					printf("%d\t",mat[i][j]);
				}
				printf("\n");
			}
  return 0;
}
